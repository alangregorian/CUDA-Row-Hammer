#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <chrono>

// Kernel to access global memory
__global__ void accessGlobalMemory(float* d_array, int stride, int accesses) {
    int idx = threadIdx.x;
    float sum = 0.0f;

    for (int i = 0; i < accesses; i++) {
        sum += d_array[(idx + i * stride) % accesses];
    }

    // Prevent compiler optimization
    if (sum > 0) {
        d_array[idx] = sum;
    }
}

void measureAccessTime(float* d_array, int stride, int accesses, std::ofstream& file) {
    auto start = std::chrono::high_resolution_clock::now();
    accessGlobalMemory<<<1, 1>>>(d_array, stride, accesses);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> diff = end - start;
    file << "Stride: " << stride << ", Time: " << diff.count() << " s\n";
}

int main() {
    const int size = 1024 * 1024; // Size of the array (1M elements)
    const int accesses = 1024; // Number of accesses to test
    float* h_array = new float[size];
    float* d_array;

    // Initialize host array
    for (int i = 0; i < size; i++) {
        h_array[i] = static_cast<float>(i);
    }

    // Allocate device memory
    hipMalloc(&d_array, size * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_array, h_array, size * sizeof(float), hipMemcpyHostToDevice);

    // Open the file to write results
    std::ofstream file("access_latency_timing.txt");

    // Test with different strides
    for (int stride = 1; stride <= 8192; stride *= 2) {
        measureAccessTime(d_array, stride, accesses, file);
    }

    // Close the file
    file.close();

    // Free device memory
    hipFree(d_array);

    // Free host memory
    delete[] h_array;

    return 0;
}
