#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstdlib>
#include <ctime>

#define N 1024*1024  // Number of iterations

__global__ void write_latency_test(volatile int* d_array, const int* d_random_values, unsigned long long* d_time) {
    for (int idx = 0; idx < 5000; idx++) {
        unsigned long long start_time = clock();

        int temp = d_random_values[idx];
        // Write operation
        d_array[idx] = temp;

        unsigned long long end_time = clock();

        // Record the latency
        d_time[idx] = end_time - start_time;
    }
}

int main() {
    srand(time(0));

    volatile int *d_array;
    int *d_random_values;
    unsigned long long *d_time;
    hipMalloc((void**)&d_array, N * sizeof(volatile int));
    hipMalloc(&d_random_values, N * sizeof(int));
    hipMalloc(&d_time, N * sizeof(unsigned long long));

    int *h_random_values = new int[N];
    for (int i = 0; i < N; i++) {
        h_random_values[i] = rand() % 1000;  // Random values between 0 and 999
    }

    // Copy random values to device
    hipMemcpy(d_random_values, h_random_values, N * sizeof(int), hipMemcpyHostToDevice);

    // Launch a single thread
    write_latency_test<<<1, 1>>>(d_array, d_random_values, d_time);
    hipDeviceSynchronize();

    // Copy the timing data back to the host
    unsigned long long *h_time = new unsigned long long[N];
    hipMemcpy(h_time, d_time, N * sizeof(unsigned long long), hipMemcpyDeviceToHost);

    // Save the timings to a CSV file
    std::ofstream csv_file("write_latencies.csv");
    csv_file << "Iteration,Time\n";
    for (int i = 0; i < 5000; i++) {
        csv_file << i << "," << h_time[i] << "\n";
    }
    csv_file.close();

    delete[] h_random_values;
    delete[] h_time;
    hipFree((void*)d_array);
    hipFree(d_random_values);
    hipFree(d_time);
    return 0;
}
