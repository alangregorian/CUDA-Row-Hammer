#include <ctime>
#include <iostream>
#include <limits>

#include <getopt.h>

#include <hip/hip_runtime.h>

const struct option LongOptions[] = {
    {"help", no_argument, nullptr, 'h'},
    {"iterations", required_argument, nullptr, 'i'},
    {"size", required_argument, nullptr, 'n'},
    {"stride", required_argument, nullptr, 's'}
};

void initPattern(unsigned int *pattern, const size_t size,
    const size_t stride);
void printResults(const unsigned int *indices, const unsigned int *latencies,
    const size_t iterations, const size_t size, const size_t stride);
void shufflePattern(unsigned int *pattern, const size_t size,
    const size_t stride);
void usage(const char *program);

__global__ void pointerChaseKernel(unsigned int *pattern,
    unsigned int *indices, unsigned int *latencies, const size_t iterations);

void initPattern(unsigned int *pattern, const size_t size,
    const size_t stride) {
    for (size_t i = 0; i < size; i += stride) {
        pattern[i] = i;
    }

    shufflePattern(pattern, size, stride);
}

void printResults(const unsigned int *indices, const unsigned int *latencies,
    const size_t iterations, const size_t size, const size_t stride) {
    std::cout << "size,stride,index,latency" << std::endl;
    for (size_t i = 0; i < iterations; i++) {
        std::cout << size << "," << stride * sizeof(unsigned int) << ","
                  << indices[i] << "," << latencies[i] << std::endl;
    }
}

// Durstenfeld shuffle
// https://en.wikipedia.org/wiki/Fisher%E2%80%93Yates_shuffle#The_modern_algorithm
void shufflePattern(unsigned int *pattern, const size_t size,
    const size_t stride) {
    for (size_t i = (size / stride) - 1; i > 0; i--) {
        unsigned int j = rand() % (i + 1);

        std::swap(pattern[j*stride], pattern[i*stride]);
    }
}

void usage(const char *program) {
    std::cout << "Usage: " << program << " [options]" << std::endl
              << "Options:" << std::endl
              << "  -h (--help)       \tShow help message" << std::endl
              << "  -i (--iterations) \tSet iterations" << std::endl
              << "  -n (--size)       \tSet array size" << std::endl
              << "  -s (--stride)     \tSet stride" << std::endl;
}

// Fine-grained pointer chasing
// https://arxiv.org/pdf/1509.02308
__global__ void pointerChaseKernel(unsigned int *pattern,
    unsigned int *indices, unsigned int *latencies, const size_t iterations) {
    clock_t  start, stop;
    unsigned int j = 0;

    for (size_t i = 0; i < iterations; i++) {
        start = clock();
        j = pattern[j];
        indices[i] = j;
        stop = clock();

        latencies[i] = stop - start;
    }
}

int main(int argc, char *argv[]) {
    size_t iterations = 0;
    size_t size = 0;
    size_t stride = 0;

    try {
        int opt, optValue;
        while ((opt = getopt_long(argc, argv, "hi:n:s:",
                LongOptions, nullptr)) != -1) {
            switch (opt) {
                case 'h':
                    usage(argv[0]);
                    return 0;
                case 'i':
                    if (optarg) {
                        optValue = std::stoi(optarg);
                        if (optValue > 0) {
                            iterations = optValue;
                        } else {
                            std::cerr << "Error: Invalid iterations value"
                                      << std::endl;
                            return -1;
                        }
                    } else {
                        usage(argv[0]);
                        return -1;
                    }
                    break;
                case 'n':
                    if (optarg) {
                        optValue = std::stoi(optarg);
                        if ((optValue > 0) && ((optValue % 2) == 0) &&
                            (optValue <
                                std::numeric_limits<unsigned int>::max())) {
                            size = optValue;
                        } else {
                            std::cerr << "Error: Invalid size value"
                                      << std::endl;
                            return -1;
                        }
                    } else {
                        usage(argv[0]);
                        return -1;
                    }
                    break;
                case 's':
                    if (optarg) {
                        optValue = std::stoi(optarg);
                        if (optValue > 0 && optValue >= sizeof(unsigned int)) {
                            stride = (pow(2, ceil(log2(optValue))) /
                                      sizeof(unsigned int));
                        } else {
                            std::cerr << "Error: Invalid stride value"
                                      << std::endl;
                            return -1;
                        }
                    } else {
                        usage(argv[0]);
                        return -1;
                    }
                    break;
                case '?':
                    usage(argv[0]);
                    return -1;
                default:
                    usage(argv[0]);
                    return -1;
            }
        }
    } catch (const std::invalid_argument &ia) {
        std::cerr << "Invalid argument: " << ia.what() << std::endl;
        return -1;
    } catch (const std::out_of_range &oor) {
        std::cerr << "Out of range error: " << oor.what() << std::endl;
        return -1;
    }

    if (iterations == 0) {
        std::cerr << "Error: No iterations value provided" << std::endl;
        return -1;
    } else if (size == 0) {
        std::cerr << "Error: No size value provided" << std::endl;
        return -1;
    } else if (stride == 0) {
        std::cerr << "Error: No stride value provided" << std::endl;
        return -1;
    }

    if (stride > size) {
        std::cerr << "Error: Stride is larger than size" << std::endl;
        return -1;
    }

    srand(std::time(nullptr));
    
    unsigned int *hostLatencies, *hostPattern, *hostIndices;
    hostIndices = (unsigned int*)malloc(iterations * sizeof(unsigned int));
    hostLatencies = (unsigned int*)malloc(iterations * sizeof(unsigned int));
    hostPattern = (unsigned int*)malloc(size * sizeof(unsigned int));

    memset(hostIndices, 0, iterations * sizeof(unsigned int));
    memset(hostLatencies, 0, iterations * sizeof(unsigned int));
    memset(hostPattern, 0, size * sizeof(unsigned int));
    initPattern(hostPattern, size, stride);

    unsigned int *deviceIndices, *deviceLatencies, *devicePattern;
    hipMalloc((void**)&deviceIndices, iterations * sizeof(unsigned int));
    hipMalloc((void**)&deviceLatencies, iterations * sizeof(unsigned int));
    hipMalloc((void**)&devicePattern, size * sizeof(unsigned int));

    hipMemset(deviceIndices, 0, iterations * sizeof(unsigned int));
    hipMemset(deviceLatencies, 0, iterations * sizeof(unsigned int));
    hipMemcpy(devicePattern, hostPattern, size * sizeof(unsigned int),
        hipMemcpyHostToDevice);

    hipError_t ret;

    dim3 blockDim(1, 1, 1);
    dim3 gridDim(1, 1, 1);

    void *args[] = {&devicePattern, &deviceIndices, &deviceLatencies,
        &iterations};

    ret = hipLaunchKernel(reinterpret_cast<const void*>((const void*)pointerChaseKernel), gridDim, blockDim,
                            args, 0, nullptr);
    hipDeviceSynchronize();

    if (ret != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(ret) << std::endl;

        hipFree(deviceIndices);
        hipFree(devicePattern);
        hipFree(deviceLatencies);
        free(hostIndices);
        free(hostPattern);
        free(hostLatencies);

        return -1;
    }

    hipMemcpy(hostIndices, deviceIndices, iterations * sizeof(unsigned int),
        hipMemcpyDeviceToHost);
    hipMemcpy(hostLatencies, deviceLatencies,
         iterations * sizeof(unsigned int), hipMemcpyDeviceToHost);
    printResults(hostIndices, hostLatencies, iterations, size, stride);

    hipFree(deviceIndices);
    hipFree(deviceLatencies);
    hipFree(devicePattern);
    free(hostIndices);
    free(hostLatencies);
    free(hostPattern);

    return 0;
}
