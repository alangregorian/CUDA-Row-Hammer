#include <hip/hip_runtime.h>
#include <iostream>

#define N 1024 * 1024  // Size of the memory allocation
#define ITERATIONS 1000000  // Number of hammering iterations
#define PATTERN 0xDEADBEEF  // Known pattern to initialize memory

__global__ void initMemoryKernel(volatile int *data, int value) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        data[idx] = value;
    }
}

__global__ void hammerKernel(volatile int *data) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = 0; i < ITERATIONS; ++i) {
        data[idx] ^= 1;  // Example operation to induce hammering
    }
}

bool checkMemory(int *data, int value) {
    bool bitFlipsDetected = false;
    for (int i = 0; i < N; ++i) {
        if (data[i] != value) {
            std::cerr << "Bit flip detected at index " << i << ": "
                      << std::hex << data[i] << std::dec << std::endl;
            bitFlipsDetected = true;
        }
    }
    return bitFlipsDetected;
}

int main() {
    // Allocate memory on the GPU
    volatile int *d_data;
    hipError_t err = hipMalloc((void**)&d_data, N * sizeof(int));
    if (err != hipSuccess) {
        std::cerr << "Failed to allocate memory on GPU: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // Initialize memory with a known pattern
    initMemoryKernel<<<(N + 255) / 256, 256>>>(d_data, PATTERN);
    hipDeviceSynchronize();

    // Perform hammering
    hammerKernel<<<(N + 255) / 256, 256>>>(d_data);
    hipDeviceSynchronize();

    // Allocate host memory to copy data back
    int *h_data = (int*)malloc(N * sizeof(int));
    if (h_data == nullptr) {
        std::cerr << "Failed to allocate host memory" << std::endl;
        hipFree((void*)d_data);
        return -1;
    }

    // Copy data back to host
    hipMemcpy(h_data, (const int*)d_data, N * sizeof(int), hipMemcpyDeviceToHost);

    // Check for bit flips
    bool success = checkMemory(h_data, PATTERN);

    if (success) {
        std::cout << "Bit flips detected!" << std::endl;
    } else {
        std::cout << "No bit flips detected." << std::endl;
    }

    // Free allocated memory
    free(h_data);
    hipFree((void*)d_data);

    return 0;
}
